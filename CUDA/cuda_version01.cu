#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <sys/time.h>
#include <math.h>
#define N 10000001
#define THREADS_PER_BLOCK 512


__global__ void assignW(double *w, double *x) {
	   double v, k, a, b, r;
	   v = 1.0;
	   k = 0.05;
	   a = 0.0;
	   b = 1.0;
	   r = v * ( b - a ) / k;
	   *w = ( 1.0 - exp ( r * *x ) ) / ( 1.0 - exp ( r ) );
}

void random_ints(int* x, int size){
	int i;
	for (i=0;i<size;i++) {
		x[i]=rand()%100;
	}
}


// Track CPU Time
double cpuSecond() {
       struct timeval tp;
       gettimeofday(&tp,NULL);
       return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}
		
int main ( );
double *r8vec_linspace_new ( int n, double a, double b );
double *trisolve ( int n, double a[], double b[] );


int main(void) {
	
	double a;
	double *a3;
	double b;
	char command_filename[] = "fd1d_advection_diffusion_steady_commands.txt";
	FILE *command_unit;
	char data_filename[] = "fd1d_advection_diffusion_steady_data.txt";
	FILE *data_unit;
	double dx;
	double *f;
	int i;
	int j;
	double k;
	int nx;
	double *u;
	double v;
	double *w;
	double *d_w;
	double *x;
	double *d_x;
	double start, end;

	printf ( "\n" );
        printf ( "FD1D_ADVECTION_DIFFUSION_STEADY:\n" );
    	printf ( "  C version\n" );
      	printf ( "\n" );
        printf ( "  Solve the 1D steady advection diffusion equation:,\n" );
	printf ( "    v du/dx - k d2u/dx2 = 0\n" );
	printf ( "  with constant, positive velocity V and diffusivity K\n" );
	printf ( "  over the interval:\n" );
	printf ( "    0.0 <= x <= 1.0\n" );
	printf ( "  with boundary conditions:\n" );
	printf ( "    u(0) = 0, u(1) = 1.\n" );
	printf ( "\n" );
	printf ( "  Use finite differences\n" );
	printf ( "   d u/dx  = (u(t,x+dx)-u(t,x-dx))/2/dx\n" );
	printf ( "   d2u/dx2 = (u(x+dx)-2u(x)+u(x-dx))/dx^2\n" );
	
	// Physical constants
	v = 1.0;
	k = 0.05;

	// Spatial discretization
	nx = 10000001;
	a = 0.0;
	b = 1.0;
	dx = ( b - a ) / ( double ) ( nx - 1 );

	// Allocate memory in the device
	hipMalloc((double **)&d_x, nx * sizeof ( double ));

	// Set up the tridiagonal linear system corresponding to the boundary conditions and advection-diffusion equation
	a3 = ( double * ) malloc ( nx * 3 * sizeof ( double ) );
	f = ( double * ) malloc ( nx * sizeof ( double ) ); 

	// Start timing
	start = cpuSecond();

	x = r8vec_linspace_new ( nx, a, b );

	// Copy input data to device
	hipMemcpy(d_x, x, nx * sizeof ( double ), hipMemcpyHostToDevice);
	
	a3[0+1*nx] = 1.0;
 	f[0] = 0.0;

	for ( i = 1; i < nx - 1; i++ ){
	    a3[i+0*nx] = - v / dx / 2.0 - k / dx / dx;
	    a3[i+1*nx] = + 2.0 * k / dx / dx;
	    a3[i+2*nx] = + v / dx / 2.0 - k / dx / dx;
	    f[i] = 0.0;
	}
	a3[nx-1+1*nx] = 1.0;
	f[nx-1] = 1.0;

	u = trisolve ( nx, a3, f );
	
	// Allocate space in host
	w = ( double * ) malloc ( nx * sizeof ( double ) );
	
	// Alloc space for device copy of w
        hipMalloc((void **)&d_w, nx * sizeof ( double ));

        // Copy input data to device
	hipMemcpy(d_w, w, nx * sizeof ( double ), hipMemcpyHostToDevice);

	// Launch add() kernel on GPU with N blocks
	assignW<<<N/THREADS_PER_BLOCK+1,THREADS_PER_BLOCK>>>(d_w, d_x);

        // Copy result back to host
	hipMemcpy(w, d_w,  nx * sizeof ( double ), hipMemcpyDeviceToHost);

	// Write data file
	data_unit = fopen ( data_filename, "wt" );
	for ( j = 0; j < nx; j++ ){
	    fprintf ( data_unit, "%g  %g  %g\n", x[j], u[j], w[j] );
	}
	fclose ( data_unit );

	end = cpuSecond();

	// Write command file
	command_unit = fopen ( command_filename, "wt" );

	fprintf ( command_unit, "set term png\n" );
	fprintf ( command_unit, "set output 'fd1d_advection_diffusion_steady.png'\n" );
	fprintf ( command_unit, "set grid\n" );
	fprintf ( command_unit, "set style data lines\n" );
	fprintf ( command_unit, "unset key\n" );
	fprintf ( command_unit, "set xlabel '<---X--->'\n" );
	fprintf ( command_unit, "set ylabel '<---U(X)--->'\n" );
        fprintf ( command_unit, "set title 'Exact: green line, Approx: red dots'\n" );
	fprintf ( command_unit, "plot '%s' using 1:2 with points pt 7 ps 2,\\\n", data_filename );
	fprintf ( command_unit, "'' using 1:3 with lines lw 3\n" );
	fprintf ( command_unit, "quit\n" );

	fclose ( command_unit );

	// Free memory
	free ( a3 );
	free ( f );
	free ( u );
	free ( w );
	free ( x );
	hipFree(d_w);

	printf("Time taken - %.8f\n", end-start);
	
	// Terminate
	return 0;

 }


double *r8vec_linspace_new ( int n, double a, double b ){
  int i;
  double *x;

  x = ( double * ) malloc ( n * sizeof ( double ) );

  if ( n == 1 ){
       x[0] = ( a + b ) / 2.0;
  }
  else{
	for ( i = 0; i < n; i++ ){
	    x[i] = ( ( double ) ( n - 1 - i ) * a
	    	     + ( double ) (         i ) * b )
		      / ( double ) ( n - 1     );
	}
  }
return x;
}



void timestamp ( ){
# define TIME_SIZE 40

  static char time_buffer[TIME_SIZE];
    const struct tm *tm;
      time_t now;

  now = time ( NULL );
    tm = localtime ( &now );

  strftime ( time_buffer, TIME_SIZE, "%d %B %Y %I:%M:%S %p", tm );

  fprintf ( stdout, "%s\n", time_buffer );

  return;
  # undef TIME_SIZE
 }


double *trisolve ( int n, double a[], double b[] ){
  int i;
  double *x;
  double xmult;
  
  // The diagonal entries can't be zero
  for ( i = 0; i < n; i++ ){
      if ( a[i+1*n] == 0.0 ){
      	fprintf ( stderr, "\n" );
	fprintf ( stderr, "TRISOLVE - Fatal error!\n" );
	fprintf ( stderr, "  A(%d,2) = 0.\n", i );
	exit ( 1 );
	}
  }

  x = ( double * ) malloc ( n * sizeof ( double ) );

  for ( i = 0; i < n; i++ ){
      x[i] = b[i];
  }

  for ( i = 1; i < n; i++ ){
      xmult = a[i+0*n] / a[i-1+1*n];
      a[i+1*n] = a[i+1*n] - xmult * a[i-1+2*n];
      x[i]   = x[i]   - xmult * x[i-1];
  }
  x[n-1] = x[n-1] / a[n-1+1*n];

  for ( i = n - 2; 0 <= i; i-- ){
      x[i] = ( x[i] - a[i+2*n] * x[i+1] ) / a[i+1*n];
  }

  return x;
 }
							      